
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

typedef unsigned int UI;

#define BLOCK_SIZE 32

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {                                          \
    hipError_t err = value;                                                \
    if (err != hipSuccess) {                                               \
        fprintf(stderr, "Error %s at line %d in file %s\n",                 \
        hipGetErrorString(err), __LINE__, __FILE__);                       \
        exit(EXIT_FAILURE);                                                 \
    }                                                                       \
}

/* Simple utility function to check for CUDA runtime errors */
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/* Host function to compute the matrix multiplication */
void matrix_mult_cpu(int *h_a, int *h_b, int *h_result, UI n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int tmp = 0.0;
            for (int k = 0; k < n; ++k) {
                tmp += h_a[i * n + k] * h_b[k * n + j];
            }
            h_result[i * n + j] = tmp;
        }
    }
}

/* Kernel that computes the matrix multiplication */
__global__ void matrix_mult_gpu(int *d_a, int *d_b, int *d_result, UI n) {
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    int tmp = 0.0;

    for (int sub = 0; sub < n / BLOCK_SIZE; ++sub) {
        tile_a[threadIdx.y][threadIdx.x] =
                d_a[row * n + sub * BLOCK_SIZE + threadIdx.x];
        tile_b[threadIdx.y][threadIdx.x] =
                d_b[(sub * BLOCK_SIZE + threadIdx.y) * n + col];
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    d_result[row * n + col] = tmp;
}

int main(int argc, char **argv) {
    UI n, mat_size;
    //scanf("%u", &n);
    n = 100;
    mat_size = n * n * sizeof(int);

    /* Code to create two events in order to compute elapsed time */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /*------------------------ COMPUTATION ON CPU ----------------------------*/
    int *h_a, *h_b, *h_result;
    hipHostMalloc((void **) &h_a, mat_size, hipHostMallocDefault);
    hipHostMalloc((void **) &h_b, mat_size, hipHostMallocDefault);
    hipHostMalloc((void **) &h_result, mat_size, hipHostMallocDefault);
    checkCUDAError("hipHostMalloc error");

    /* Initialize input matrixs */
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
            h_b[i * n + j] = rand() % 1024;
        }
    }

    // cudaEventRecord(start, 0);

    // /* Compute matrix multiplication on CPU (host) */
    // matrix_mult_cpu(h_a, h_b, h_result, n);

    // /* Compute the host elapsed time */
    // cudaEventRecord(stop, 0);
    // cudaEventSynchronize(stop);
    // float elapsed_time_cpu;
    // cudaEventElapsedTime(&elapsed_time_cpu, start, stop);

    // printf("CPU matrix multiplication has finished!\nElapsed time: %f ms\n\n",
    //                 elapsed_time_cpu);

    /*------------------------ COMPUTATION ON GPU ----------------------------*/

    /* Allocate memory space on the device */
    int *d_a, *d_b, *d_result, *h_result_gpu;
    hipMalloc((void **) &d_a, mat_size);
    hipMalloc((void **) &d_b, mat_size);
    hipMalloc((void **) &d_result, mat_size);
    checkCUDAError("hipMalloc error");

    hipHostMalloc((void **) &h_result_gpu, mat_size, hipHostMallocDefault);
    checkCUDAError("hipHostMalloc for h_result_gpu error");

    /* Transfer data from host to device */
    hipMemcpy(d_a, h_a, mat_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, mat_size, hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy (from host to device) error");

    /* Execution configuration setup */
    /* Note: ceil(n / BLOCK_SIZE) also works for dim_grid setup */
    dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

    hipEventRecord(start, 0);

    /* Compute matrix multiplication on GPU (device kernel launch) */
    matrix_mult_gpu<<<dim_grid, dim_block>>>(d_a, d_b, d_result, n);

    /* Block until the device has completed */
    //cudaDeviceSynchronize();

    /* Compute the device elapsed time */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed_time_gpu;
    hipEventElapsedTime(&elapsed_time_gpu, start, stop);

    printf("GPU matrix multiplication has finished!\nElapsed time: %f ms\n\n",
                elapsed_time_gpu);

    /* Transefr results from device to host */
    hipMemcpy(h_result_gpu, d_result, mat_size, hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy (from device to host) error");

    /* Compare against host computed solution (no valid for floating point) */
    // for (int i = 0; i < n; ++i) {
    //     for (int j = 0; j < n; ++j) {
    //         assert(h_result[i * n + j] == h_result_gpu[i * n + j]);
    //     }
    // }

    printf("Matrix Multiplication on both CPU and GPU are correct!\n\n");

    /* Destroy CUDA Events */
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* Free device memory */
    hipFree(d_result);
    hipFree(d_a);
    hipFree(d_b);
    checkCUDAError("hipFree error");

    /* Free host memory (it also works with free() call) */
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_result);
    hipHostFree(h_result_gpu);
    checkCUDAError("hipHostFree error");

    // printf("Speedup of GPU version over the CPU version for an %u x %u input "
    //         "matrixs is %fX\n", n, n, elapsed_time_cpu / elapsed_time_gpu);
    printf("Time of GPU version for an %u x %u input "
            "matrixs is %fX\n", n, n, elapsed_time_gpu);
}

